#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define SIZEOFINT sizeof(int)

#define TH_DIM 32
const dim3 thread_dim(TH_DIM, TH_DIM);
const int block_num = 5000;

const int INF = ((1 << 30) - 1);
const int V = 4000;
void show_mat(int *, int);
void malloc_Dist(int);
int getDist(int ,int, int);
int *getDistAddr(int ,int, int);
void setDist(int ,int, int, int);
void input(char* inFileName);
void output(char* outFileName);

void block_FW(int B);
int ceil(int a, int b);
void cal(int vertex_num, int edge_num, int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height);

int n, m;
int *Dist;
// int Dist[V][V];
int *cuda_Dist;

int main(int argc, char* argv[]) {
    input(argv[1]);
    // show_mat(getDistAddr(0, 0, n), n);
    int B = 512;
    printf("Vertice: %d, Edge: %d\n", n, m);
    block_FW(B);
    // show_mat(getDistAddr(0, 0, n), n);
    output(argv[2]);
    // show_mat(getDistAddr(0, 0, n), n);
    return 0;
}
void show_mat(int *start_p, int vertex_num){
    for(int i = 0; i < vertex_num; i++){
        for(int j = 0; j < vertex_num; j++){
            printf("%d\t  ", start_p[i * vertex_num + j]);
        }
        printf("\n");
    }
}
void malloc_Dist(int vertex_num){Dist = (int*)malloc(SIZEOFINT * vertex_num * vertex_num);}
int getDist(int i, int j, int vertex_num){return Dist[i * vertex_num + j];}
int *getDistAddr(int i, int j, int vertex_num){return &(Dist[i * vertex_num + j]);}
void setDist(int i, int j, int val, int vertex_num){Dist[i * vertex_num + j] = val;}

void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);
    malloc_Dist(n);
    hipMalloc((void **)&cuda_Dist, (n * n * SIZEOFINT));

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i == j) {
                setDist(i, j, 0, n);
                // Dist[i][j] = 0;
            } else {
                setDist(i, j, INF, n);
                // Dist[i][j] = INF;
            }
        }
    }

    int pair[3];
    for (int i = 0; i < m; i++) {
        fread(pair, sizeof(int), 3, file);
        setDist(pair[0], pair[1], pair[2], n);
        // Dist[pair[0]][pair[1]] = pair[2];
    }
    hipMemcpy(cuda_Dist, Dist, (n * n * SIZEOFINT), hipMemcpyHostToDevice);
    fclose(file);
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            // if (Dist[i][j] >= INF) Dist[i][j] = INF;
            if (getDist(i, j, n) >= INF) setDist(i, j, INF, n);
        }
        // fwrite(Dist[i], sizeof(int), n, outfile);
        // fwrite(getDistAddr(i, 0, n), sizeof(int), n, outfile);
    }
    fwrite(getDistAddr(0, 0, n), sizeof(int), n * n, outfile);
    fclose(outfile);
}

int ceil(int a, int b) { return (a + b - 1) / b; }

void block_FW(int B) {
    int round = ceil(n, B);
    for (int r = 0; r < round; r++) {
        printf("Round: %d in total: %d\n", r, round);
        fflush(stdout);
        /* Phase 1*/
        cal(n, m, B, r, r, r, 1, 1);

        /* Phase 2*/
        cal(n, m, B, r, r, 0, r, 1);
        cal(n, m, B, r, r, r + 1, round - r - 1, 1);
        cal(n, m, B, r, 0, r, 1, r);
        cal(n, m, B, r, r + 1, r, 1, round - r - 1);

        /* Phase 3*/
        cal(n, m, B, r, 0, 0, r, r);
        cal(n, m, B, r, 0, r + 1, round - r - 1, r);
        cal(n, m, B, r, r + 1, 0, r, round - r - 1);
        cal(n, m, B, r, r + 1, r + 1, round - r - 1, round - r - 1);
    }
}

void cal(int vertex_num, int edge_num, int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height) {
    int block_end_x = block_start_x + block_height;
    int block_end_y = block_start_y + block_width;

    for (int b_i = block_start_x; b_i < block_end_x; b_i++) {
        for (int b_j = block_start_y; b_j < block_end_y; b_j++) {
            // To calculate B*B elements in the block (b_i, b_j)
            // For each block, it need to compute B times
            for (int k = Round * B; k < (Round + 1) * B && k < vertex_num; k++) {
                // To calculate original index of elements in the block (b_i, b_j)
                // For instance, original index of (0,0) in block (1,2) is (2,5) for V=6,B=2
                int block_internal_start_x = b_i * B;
                int block_internal_end_x = (b_i + 1) * B;
                int block_internal_start_y = b_j * B;
                int block_internal_end_y = (b_j + 1) * B;

                if (block_internal_end_x > vertex_num) block_internal_end_x = vertex_num;
                if (block_internal_end_y > vertex_num) block_internal_end_y = vertex_num;

                for (int i = block_internal_start_x; i < block_internal_end_x; i++) {
                    for (int j = block_internal_start_y; j < block_internal_end_y; j++) {
                        // if (Dist[i][k] + Dist[k][j] < Dist[i][j]) {
                        //     Dist[i][j] = Dist[i][k] + Dist[k][j];
                        // }
                        if (getDist(i, k, vertex_num) + getDist(k, j, vertex_num) < getDist(i, j, vertex_num)) {
                            setDist(i, j, getDist(i, k, vertex_num) + getDist(k, j, vertex_num), vertex_num);
                        }
                    }
                }
            }
        }
    }
}
