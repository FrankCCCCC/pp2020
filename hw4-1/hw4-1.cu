#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define SIZEOFINT sizeof(int)

#define TH_DIM 32
const dim3 thread_dim(TH_DIM, TH_DIM);
const int block_num = 5000;

const int INF = ((1 << 30) - 1);
const int V = 4000;
// void show_mat(int *, int);
// void malloc_Dist(int);
// void setup_DistCuda(int);
// int getDist(int ,int, int);
// int *getDistAddr(int ,int, int);
// void setDist(int ,int, int, int);
// void input(char* inFileName);
// void output(char* outFileName);

// void block_FW_cuda(int B);
// __global__ void cal_cuda(int *Dist, int vertex_num, int edge_num, int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height);
// void block_FW(int B);
// int ceil(int a, int b);
// void cal(int vertex_num, int edge_num, int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height);

const int B = 32;
int n, m;
int *Dist;
int *Dist_cuda;

void show_mat(int *start_p, int vertex_num){
    for(int i = 0; i < vertex_num; i++){
        for(int j = 0; j < vertex_num; j++){
            if(start_p[i * vertex_num + j] == INF){
                printf("INF\t  ");
            }else{
                printf("%d\t  ", start_p[i * vertex_num + j]);
            }
            
        }
        printf("\n");
    }
}

void malloc_Dist(int vertex_num){Dist = (int*)malloc(SIZEOFINT * vertex_num * vertex_num);}
int getDist(int i, int j, int vertex_num){return Dist[i * vertex_num + j];}
int *getDistAddr(int i, int j, int vertex_num){return &(Dist[i * vertex_num + j]);}
void setDist(int i, int j, int val, int vertex_num){Dist[i * vertex_num + j] = val;}

void setup_DistCuda(int vertex_num){
    hipMalloc((void **)&Dist_cuda, SIZEOFINT * vertex_num * vertex_num);
    hipMemcpy(Dist_cuda, Dist, (n * n * SIZEOFINT), hipMemcpyHostToDevice);
}
void back_DistCuda(int vertex_num){
    hipMemcpy(Dist, Dist_cuda, (n * n * SIZEOFINT), hipMemcpyDeviceToHost);
}
// int getDistCuda(int i, int j, int vertex_num){return Dist_cuda[i * vertex_num + j];}
// int *getDistAddrCuda(int i, int j, int vertex_num){return &(Dist_cuda[i * vertex_num + j]);}
// void setDistCuda(int i, int j, int val, int vertex_num){Dist_cuda[i * vertex_num + j] = val;}

void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);
    malloc_Dist(n);
    // malloc_DistCuda(n);

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i == j) {
                setDist(i, j, 0, n);
                // Dist[i][j] = 0;
            } else {
                setDist(i, j, INF, n);
                // Dist[i][j] = INF;
            }
        }
    }

    int pair[3];
    for (int i = 0; i < m; i++) {
        fread(pair, sizeof(int), 3, file);
        setDist(pair[0], pair[1], pair[2], n);
        // Dist[pair[0]][pair[1]] = pair[2];
    }
    // cudaMemcpy(Dist_cuda, Dist, (n * n * SIZEOFINT), cudaMemcpyHostToDevice);
    fclose(file);
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            // if (Dist[i][j] >= INF) Dist[i][j] = INF;
            if (getDist(i, j, n) >= INF) setDist(i, j, INF, n);
        }
        // fwrite(Dist[i], sizeof(int), n, outfile);
        // fwrite(getDistAddr(i, 0, n), sizeof(int), n, outfile);
    }
    fwrite(getDistAddr(0, 0, n), sizeof(int), n * n, outfile);
    fclose(outfile);
}

__global__ void cal_cuda(int *dist, int vertex_num, int edge_num, int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height) {
    int block_end_x = block_start_x + block_height;
    int block_end_y = block_start_y + block_width;
    __shared__ int s_m[100][100];
    // printf("%d\n", dist[1]);

    for (int b_i = block_start_x; b_i < block_end_x; b_i++) {
        for (int b_j = block_start_y; b_j < block_end_y; b_j++) {
            // To calculate B*B elements in the block (b_i, b_j)
            // For each block, it need to compute B times

            // for (int i = block_internal_start_x + threadIdx.x; i < block_internal_end_x; i+=blockDim.x) {
            //     for (int j = block_internal_start_y + threadIdx.y; j < block_internal_end_y; j+=blockDim.y) {
                    
            //     }
            // }

            for (int k = Round * B; k < (Round + 1) * B && k < vertex_num; k++) {
                // To calculate original index of elements in the block (b_i, b_j)
                // For instance, original index of (0,0) in block (1,2) is (2,5) for V=6,B=2
                int block_internal_start_x = b_i * B;
                int block_internal_end_x = (b_i + 1) * B;
                int block_internal_start_y = b_j * B;
                int block_internal_end_y = (b_j + 1) * B;

                if (block_internal_end_x > vertex_num) block_internal_end_x = vertex_num;
                if (block_internal_end_y > vertex_num) block_internal_end_y = vertex_num;

                for (int i = block_internal_start_x + threadIdx.x; i < block_internal_end_x; i+=blockDim.x) {
                    for (int j = block_internal_start_y + threadIdx.y; j < block_internal_end_y; j+=blockDim.y) {
                        int d = dist[i * vertex_num + k] + dist[k * vertex_num + j];
                        if (d < dist[i * vertex_num + j]) {
                            dist[i * vertex_num + j] = d;
                        }
                        // dist[i * vertex_num + j] = 2;
                    }
                }
                __syncthreads();
            }
        }
    }
}

void block_FW_cuda(int B) {
    int round = (n + B - 1) / B;
    for (int r = 0; r < round; r++) {
        printf("Round: %d in total: %d\n", r, round);
        fflush(stdout);
        /* Phase 1*/
        cal_cuda<<<1, thread_dim>>>(Dist_cuda, n, m, B, r, r, r, 1, 1);

        /* Phase 2*/
        cal_cuda<<<1, thread_dim>>>(Dist_cuda, n, m, B, r, r, 0, r, 1);
        cal_cuda<<<1, thread_dim>>>(Dist_cuda, n, m, B, r, r, r + 1, round - r - 1, 1);
        cal_cuda<<<1, thread_dim>>>(Dist_cuda, n, m, B, r, 0, r, 1, r);
        cal_cuda<<<1, thread_dim>>>(Dist_cuda, n, m, B, r, r + 1, r, 1, round - r - 1);

        /* Phase 3*/
        cal_cuda<<<1, thread_dim>>>(Dist_cuda, n, m, B, r, 0, 0, r, r);
        cal_cuda<<<1, thread_dim>>>(Dist_cuda, n, m, B, r, 0, r + 1, round - r - 1, r);
        cal_cuda<<<1, thread_dim>>>(Dist_cuda, n, m, B, r, r + 1, 0, r, round - r - 1);
        cal_cuda<<<1, thread_dim>>>(Dist_cuda, n, m, B, r, r + 1, r + 1, round - r - 1, round - r - 1);
    }
}

void cal(int vertex_num, int edge_num, int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height) {
    int block_end_x = block_start_x + block_height;
    int block_end_y = block_start_y + block_width;

    for (int b_i = block_start_x; b_i < block_end_x; b_i++) {
        for (int b_j = block_start_y; b_j < block_end_y; b_j++) {
            // To calculate B*B elements in the block (b_i, b_j)
            // For each block, it need to compute B times            

            for (int k = Round * B; k < (Round + 1) * B && k < vertex_num; k++) {
                // To calculate original index of elements in the block (b_i, b_j)
                // For instance, original index of (0,0) in block (1,2) is (2,5) for V=6,B=2
                int block_internal_start_x = b_i * B;
                int block_internal_end_x = (b_i + 1) * B;
                int block_internal_start_y = b_j * B;
                int block_internal_end_y = (b_j + 1) * B;

                if (block_internal_end_x > vertex_num) block_internal_end_x = vertex_num;
                if (block_internal_end_y > vertex_num) block_internal_end_y = vertex_num;

                for (int i = block_internal_start_x; i < block_internal_end_x; i++) {
                    for (int j = block_internal_start_y; j < block_internal_end_y; j++) {
                        // if (Dist[i][k] + Dist[k][j] < Dist[i][j]) {
                        //     Dist[i][j] = Dist[i][k] + Dist[k][j];
                        // }
                        if (getDist(i, k, vertex_num) + getDist(k, j, vertex_num) < getDist(i, j, vertex_num)) {
                            setDist(i, j, getDist(i, k, vertex_num) + getDist(k, j, vertex_num), vertex_num);
                        }
                    }
                }
            }
        }
    }
}

int ceil(int a, int b) { return (a + b - 1) / b; }

void block_FW(int B) {
    int round = ceil(n, B);
    for (int r = 0; r < round; r++) {
        printf("Round: %d in total: %d\n", r, round);
        fflush(stdout);
        /* Phase 1*/
        cal(n, m, B, r, r, r, 1, 1);

        /* Phase 2*/
        cal(n, m, B, r, r, 0, r, 1);
        cal(n, m, B, r, r, r + 1, round - r - 1, 1);
        cal(n, m, B, r, 0, r, 1, r);
        cal(n, m, B, r, r + 1, r, 1, round - r - 1);

        /* Phase 3*/
        cal(n, m, B, r, 0, 0, r, r);
        cal(n, m, B, r, 0, r + 1, round - r - 1, r);
        cal(n, m, B, r, r + 1, 0, r, round - r - 1);
        cal(n, m, B, r, r + 1, r + 1, round - r - 1, round - r - 1);
    }
}



int main(int argc, char* argv[]) {
    input(argv[1]);
    // show_mat(getDistAddr(0, 0, n), n);
    setup_DistCuda(n);
    printf("Vertice: %d, Edge: %d\n", n, m);
    // block_FW(B);
    block_FW_cuda(B);
    back_DistCuda(n);
    // show_mat(getDistAddr(0, 0, n), n);
    
    output(argv[2]);
    // show_mat(getDistAddr(0, 0, n), n);
    return 0;
}