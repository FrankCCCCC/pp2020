#include <iostream>
#include <algorithm>
#include <stdio.h>

#include <hip/hip_runtime.h>

#include <driver_functions.h>

// #include "floyd_warshall.hpp"

#define BLOCK_DIM 16
#define SIZEOFINT sizeof(int)
const int INF = ((1 << 30) - 1);
// const int B = 64;
int n, m;
int *Dist;
int *Dist_out;
// int *Dist_cuda;

__forceinline__
__host__ void check_cuda_error() {
  hipError_t errCode = hipPeekAtLastError();
  if (errCode != hipSuccess) {
    std::cerr << "WARNING: A CUDA error occured: code=" << errCode << "," <<
                hipGetErrorString(errCode) << "\n";
  }
}

__forceinline__
__device__ void calc(int* graph, int n, int k, int i, int j) {
  if ((i >= n) || (j >= n) || (k >= n)) return;
  const unsigned int kj = k*n + j;
  const unsigned int ij = i*n + j;
  const unsigned int ik = i*n + k;
  int t1 = graph[ik] + graph[kj];
  int t2 = graph[ij];
  graph[ij] = (t1 < t2) ? t1 : t2;
}


__global__ void floyd_warshall_kernel(int n, int k, int* graph) {
  const unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
  const unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
  calc(graph, n, k, i, j);
}

/*****************************************************************************
                         Blocked Floyd-Warshall Kernel
  ***************************************************************************/

__forceinline__
__device__ void block_calc(int* C, int* A, int* B, int bj, int bi) {
  for (int k = 0; k < BLOCK_DIM; k++) {
    int sum = A[bi*BLOCK_DIM + k] + B[k*BLOCK_DIM + bj];
    if (C[bi*BLOCK_DIM + bj] > sum) {
      C[bi*BLOCK_DIM + bj] = sum;
    }
    __syncthreads();
  }
}

__global__ void floyd_warshall_block_kernel_phase1(int n, int k, int* graph) {
  const unsigned int bi = threadIdx.y;
  const unsigned int bj = threadIdx.x;

  __shared__ int C[BLOCK_DIM * BLOCK_DIM];

  __syncthreads();

  // Transfer to temp shared arrays
  C[bi*BLOCK_DIM + bj] = graph[k*BLOCK_DIM*n + k*BLOCK_DIM + bi*n + bj];

  __syncthreads();
  
  block_calc(C, C, C, bi, bj);

  __syncthreads();

  // Transfer back to graph
  graph[k*BLOCK_DIM*n + k*BLOCK_DIM + bi*n + bj] = C[bi*BLOCK_DIM + bj];

}


__global__ void floyd_warshall_block_kernel_phase2(int n, int k, int* graph) {
  // BlockDim is one dimensional (Straight along diagonal)
  // Blocks themselves are two dimensional
  const unsigned int i = blockIdx.x;
  const unsigned int bi = threadIdx.y;
  const unsigned int bj = threadIdx.x;

  if (i == k) return;

  __shared__ int A[BLOCK_DIM * BLOCK_DIM];
  __shared__ int B[BLOCK_DIM * BLOCK_DIM];
  __shared__ int C[BLOCK_DIM * BLOCK_DIM];

  __syncthreads();

  C[bi*BLOCK_DIM + bj] = graph[i*BLOCK_DIM*n + k*BLOCK_DIM + bi*n + bj];
  B[bi*BLOCK_DIM + bj] = graph[k*BLOCK_DIM*n + k*BLOCK_DIM + bi*n + bj];

  __syncthreads();

  block_calc(C, C, B, bi, bj);

  __syncthreads();

  graph[i*BLOCK_DIM*n + k*BLOCK_DIM + bi*n + bj] = C[bi*BLOCK_DIM + bj];

  // Phase 2 1/2

  C[bi*BLOCK_DIM + bj] = graph[k*BLOCK_DIM*n + i*BLOCK_DIM + bi*n + bj];
  A[bi*BLOCK_DIM + bj] = graph[k*BLOCK_DIM*n + k*BLOCK_DIM + bi*n + bj];

  __syncthreads();

  block_calc(C, A, C, bi, bj);

  __syncthreads();

  // Block C is the only one that could be changed
  graph[k*BLOCK_DIM*n + i*BLOCK_DIM + bi*n + bj] = C[bi*BLOCK_DIM + bj];
}


__global__ void floyd_warshall_block_kernel_phase3(int n, int k, int* graph) {
  // BlockDim is one dimensional (Straight along diagonal)
  // Blocks themselves are two dimensional
  const unsigned int j = blockIdx.x;
  const unsigned int i = blockIdx.y;
  const unsigned int bi = threadIdx.y;
  const unsigned int bj = threadIdx.x;

  if (i == k && j == k) return;
  __shared__ int A[BLOCK_DIM * BLOCK_DIM];
  __shared__ int B[BLOCK_DIM * BLOCK_DIM];
  __shared__ int C[BLOCK_DIM * BLOCK_DIM];

  __syncthreads();

  C[bi*BLOCK_DIM + bj] = graph[i*BLOCK_DIM*n + j*BLOCK_DIM + bi*n + bj];
  A[bi*BLOCK_DIM + bj] = graph[i*BLOCK_DIM*n + k*BLOCK_DIM + bi*n + bj];
  B[bi*BLOCK_DIM + bj] = graph[k*BLOCK_DIM*n + j*BLOCK_DIM + bi*n + bj];

  __syncthreads();

  block_calc(C, A, B, bi, bj);

  __syncthreads();

  graph[i*BLOCK_DIM*n + j*BLOCK_DIM + bi*n + bj] = C[bi*BLOCK_DIM + bj];
}

/************************************************************************
                    Floyd-Warshall's Algorithm CUDA
************************************************************************/


__host__ void floyd_warshall_blocked_cuda(int* input, int* output, int n) {

  int deviceCount;
  hipGetDeviceCount(&deviceCount);

  for (int i = 0; i < deviceCount; i++) {
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, i);

    std::cout << "Device " << i << ": " << deviceProps.name << "\n"
	      << "\tSMs: " << deviceProps.multiProcessorCount << "\n"
	      << "\tGlobal mem: " << static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024 * 1024) << "GB \n"
	      << "\tCUDA Cap: " << deviceProps.major << "." << deviceProps.minor << "\n";
  }

  int* device_graph;
  const size_t size = sizeof(int) * n * n;
  hipMalloc(&device_graph, size);
  hipMemcpy(device_graph, input, size, hipMemcpyHostToDevice);

  const int blocks = (n + BLOCK_DIM - 1) / BLOCK_DIM;
  dim3 block_dim(BLOCK_DIM, BLOCK_DIM, 1);
  dim3 phase4_grid(blocks, blocks, 1);

  std::cout << "Launching Kernels Blocks: " << blocks << " Size " << n << "\n";
  for (int k = 0; k < blocks; k++) {
    floyd_warshall_block_kernel_phase1<<<1, block_dim>>>(n, k, device_graph);

    floyd_warshall_block_kernel_phase2<<<blocks, block_dim>>>(n, k, device_graph);

    floyd_warshall_block_kernel_phase3<<<phase4_grid, block_dim>>>(n, k, device_graph);
  }
  
  hipMemcpy(output, device_graph, size, hipMemcpyDeviceToHost);
  check_cuda_error();

  hipFree(device_graph);
}

void malloc_Dist(int vertex_num){
  Dist = (int*)malloc(SIZEOFINT * vertex_num * vertex_num);
  Dist_out = (int*)malloc(SIZEOFINT * vertex_num * vertex_num);
}
int getDist(int i, int j, int vertex_num){return Dist[i * vertex_num + j];}
int *getDistAddr(int i, int j, int vertex_num){return &(Dist[i * vertex_num + j]);}
void setDist(int i, int j, int val, int vertex_num){Dist[i * vertex_num + j] = val;}

void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);
    malloc_Dist(n);
    // malloc_DistCuda(n);

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i == j) {
                setDist(i, j, 0, n);
                // Dist[i][j] = 0;
            } else {
                setDist(i, j, INF, n);
                // Dist[i][j] = INF;
            }
        }
    }

    int pair[3];
    for (int i = 0; i < m; i++) {
        fread(pair, sizeof(int), 3, file);
        setDist(pair[0], pair[1], pair[2], n);
        // Dist[pair[0]][pair[1]] = pair[2];
    }
    // cudaMemcpy(Dist_cuda, Dist, (n * n * SIZEOFINT), cudaMemcpyHostToDevice);
    fclose(file);
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            // if (Dist[i][j] >= INF) Dist[i][j] = INF;
            if (getDist(i, j, n) >= INF) setDist(i, j, INF, n);
        }
        // fwrite(Dist[i], sizeof(int), n, outfile);
        // fwrite(getDistAddr(i, 0, n), sizeof(int), n, outfile);
    }
    fwrite(getDistAddr(0, 0, n), sizeof(int), n * n, outfile);
    fclose(outfile);
}

int main(int argc, char* argv[]) {
    input(argv[1]);
    // show_mat(getDistAddr(0, 0, n), n);
    // setup_DistCuda(n);
    printf("Vertice: %d, Edge: %d\n", n, m);
    floyd_warshall_blocked_cuda(Dist, Dist_out, n);
    // block_FW_cuda(B);
    // back_DistCuda(n);
    // show_mat(getDistAddr(0, 0, n), n);
    
    output(argv[2]);
    // show_mat(getDistAddr(0, 0, n), n);
    return 0;
}